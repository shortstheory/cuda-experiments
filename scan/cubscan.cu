#include <hipcub/hipcub.hpp>
#include <iostream>
#include <vector>

int main(int argc, char **argv) {
    int N=1000;

   if (argc > 1)
   {
       N = std::stoi(argv[1]);
   }

    std::vector<float> h_input(N), h_output(N);

    // Fill input data (e.g., with 1s or i+1)
    for (int i = 0; i < N; ++i) h_input[i] = i;

    float* d_input;
    float* d_output;
    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;

    // Allocate device memory
    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));

    // Copy input to device
    hipMemcpy(d_input, h_input.data(), N * sizeof(float), hipMemcpyHostToDevice);

    // First call to determine temp storage size
    hipcub::DeviceScan::InclusiveSum(
        nullptr, temp_storage_bytes,
        d_input, d_output, N);

    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // Second call to actually run the scan
    hipcub::DeviceScan::InclusiveSum(
        d_temp_storage, temp_storage_bytes,
        d_input, d_output, N);

    // Copy output back to host
    hipMemcpy(h_output.data(), d_output, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print output
    // std::cout << "Scan result:\n";
    // for (int i = 0; i < N; ++i) {
    //     std::cout << h_output[i] << " ";
    // }
    // std::cout << std::endl;

    // Cleanup
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_temp_storage);

    return 0;
}