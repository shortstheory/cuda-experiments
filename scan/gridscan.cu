#include "hip/hip_runtime.h"
#include <math.h>


#include <cub/block/block_load.cuh>
#include <hipcub/hipcub.hpp>
#include <cstdlib>
#include <ctime>
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include <string>  // Required for std::stoi()
constexpr int TILE = 512;


namespace cg = cooperative_groups;
using namespace cooperative_groups; // or...
using cooperative_groups::thread_group; // etc.
constexpr int PARTITION_SIZE = 32;
constexpr int BLOCK_SIZE = PARTITION_SIZE*4;


enum BlockState {
   BLOCK_WAIT=0,
   BLOCK_SUM_READY=1,
   BLOCK_SUM_DONE=2
};


__global__ void myShuffleScan(float *data, int* blockCounter, BlockState* blockStates, float* blockResults, int n) {
   __shared__ int sBlockNum;
   __shared__ int sPrefixSumIndex;
   if (threadIdx.x == 0)
   {
       sBlockNum = atomicAdd(blockCounter, 1);
       sPrefixSumIndex = max(sBlockNum-1,0);
   }
   __syncthreads();
   int offset = blockIdx.x * blockDim.x;
   int offset1 = sBlockNum * blockDim.x;

   if (threadIdx.x == 0)
   {
        printf("Offset %d offset1 %d\n",offset,offset1);
   }
   data = data + offset1;


   __shared__ float sharedData[BLOCK_SIZE/PARTITION_SIZE];


   int idx = threadIdx.x;
   float threadValue = 0.f;
   if (idx < n)
   {
       threadValue = data[idx];
   }


   auto tile = cg::tiled_partition<PARTITION_SIZE>(this_thread_block());


   #pragma unroll
   for (int i = 1; i < PARTITION_SIZE; i *= 2) {
       float temp = tile.shfl_up(threadValue, i);
       if (idx % PARTITION_SIZE >= i)
       {
           threadValue += temp;
       }
   }
   if (tile.thread_rank()+1 == PARTITION_SIZE)
   {
       sharedData[idx/PARTITION_SIZE] = threadValue;
   }
   __syncthreads();


   if (idx < PARTITION_SIZE)
   {
       #pragma unroll
       for (int i = 1; i < PARTITION_SIZE; i *= 2) {
           float temp = tile.shfl_up(sharedData[tile.thread_rank()], i);
           if (tile.thread_rank() % PARTITION_SIZE >= i)
           {
               sharedData[tile.thread_rank()] += temp;
           }
       }
   }
   __syncthreads();


   if (idx >= PARTITION_SIZE)
   {
       threadValue += sharedData[idx/PARTITION_SIZE-1];
   }

/*
   if ((idx+1) % BLOCK_SIZE == 0 || idx + 1 == n)
   {
       if (sBlockNum > 0)
       {
         blockStates[sBlockNum] = BLOCK_SUM_READY;
         blockResults[sBlockNum] = threadValue;
       }
       else
       {
           blockStates[0] = BLOCK_SUM_DONE;
           blockResults[0] = threadValue;
       }
         printf("Index %d Val %f\n", sBlockNum, blockResults[sBlockNum]);
   }


   __shared__ float sPrefixSum;
   if (threadIdx.x == 0)
   {
     sPrefixSum = 0.f;
     while (blockStates[sPrefixSumIndex] != BLOCK_SUM_DONE) {
       if (blockStates[sPrefixSumIndex] == BLOCK_SUM_READY) {
         sPrefixSum += blockResults[sPrefixSumIndex];
       }
       sPrefixSumIndex--;
     }
     sPrefixSum += blockResults[sPrefixSumIndex];
     blockStates[sBlockNum] = BLOCK_SUM_DONE;
     printf("Index %d Val %f\n", sBlockNum, sPrefixSum);
   }
*/

   if (idx < n)
   {
       data[idx] = threadValue;//+sPrefixSum;
   }
}




int main(int argc, char **argv)
{
   int n = 128;


   if (argc > 1)
   {
       n = std::stoi(argv[1]);
   }
   int numBlocks = (n + BLOCK_SIZE -1) / BLOCK_SIZE;
   std::cout << "N: " << n << std::endl;
   float* data = new float[n];
   float* gpuResultCpu = new float[n];
   float* gpuData;
   int* blockCounterGpu;
   float* blockResultGpu;
   BlockState* blockStatesGpu;
   hipError_t error = hipMalloc(&gpuData, n*sizeof(float));
    hipMalloc(&blockStatesGpu, numBlocks * sizeof(BlockState));
    hipMalloc(&blockResultGpu, numBlocks * sizeof(float));
    hipMalloc(&blockCounterGpu, sizeof(int));
   hipMemset(blockStatesGpu, 0, numBlocks * sizeof(BlockState));
   hipMemset(blockResultGpu, 0, numBlocks * sizeof(float));
   hipMemset(blockCounterGpu, 0, sizeof(int));


   std::cout << hipGetErrorString(error) << std::endl;
   for (int i = 0; i < n; i++)
   {
       data[i] = i;
   }


   hipMemcpy(gpuData, data, n*sizeof(float), hipMemcpyHostToDevice);
   myShuffleScan<<<numBlocks, BLOCK_SIZE>>>(gpuData, blockCounterGpu, blockStatesGpu,
                                      blockResultGpu, n);


   hipMemcpy(gpuResultCpu, gpuData, n*sizeof(float), hipMemcpyDeviceToHost);


   for (int i = 1; i < n; i++)
   {
       data[i] = data[i] + data[i-1];
   }
   for (int i = 0; i < n; i++)
   {
       std::cout << i << " " << data[i] << " " << gpuResultCpu[i] << std::endl;
   }
   // std::cout << std::endl;
   // for (int i = 0; i < n; i++)
   // {
   //     std::cout << i << " " <<  << " " << std::endl;
   // }


   return 0;
}

