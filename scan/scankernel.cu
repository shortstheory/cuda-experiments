#include "hip/hip_runtime.h"
#include <math.h>

#include <cub/block/block_load.cuh>
#include <hipcub/hipcub.hpp>
#include <cstdlib>
#include <ctime>
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include <string>  // Required for std::stoi()
constexpr int TILE = 512;

namespace cg = cooperative_groups;
using namespace cooperative_groups; // or...
using cooperative_groups::thread_group; // etc.
constexpr int PARTITION_SIZE = 32;

__global__ void myBlockScan(float *data, int n) {
    __shared__ float sharedData[1024];
    int idx = threadIdx.x;
    sharedData[idx] = 0.f;
    if (idx < n)
    {
        sharedData[idx] = data[idx];
    }
    __syncthreads();

    for (int i = 1; i < n; i*=2)
    {
        if (idx > i)
        {
            sharedData[idx] = sharedData[idx] + sharedData[idx-i];
        }
        __syncthreads();
    }

    if (idx < n)
    {
        data[idx] = sharedData[idx];
    }
}


int main(int argc, char **argv)
{
    int n = 1000;
    if (argc > 1) 
    {
        n = std::stoi(argv[1]);
    }
    std::cout << "N: " << n << std::endl;
    float* data = new float[n];
    float* gpuResultCpu = new float[n];
    float* gpuData;
    hipError_t error = hipMalloc(&gpuData, n*sizeof(float));
    std::cout << hipGetErrorString(error) << std::endl;
    for (int i = 0; i < n; i++)
    {
        data[i] = i;
    }

    hipMemcpy(gpuData, data, n*sizeof(float), hipMemcpyHostToDevice);
    myBlockScan<<<1,1024>>>(gpuData,n);
    hipMemcpy(gpuResultCpu, gpuData, n*sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 1; i < n; i++)
    {
        data[i] = data[i] + data[i-1];
    }
    for (int i = 0; i < n; i++)
    {
        std::cout << data[i] << " ";
    }
    std::cout << std::endl;
    for (int i = 0; i < n; i++)
    {
        std::cout << gpuResultCpu[i] << " ";
    }

    return 0;
}